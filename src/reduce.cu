#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"

constexpr inline bool is_power_of_2(int n) {
    return (n & (n-1)) == 0;
}

template <uint warpSize, typename T, typename OpFn>
__device__
__forceinline__
T reduce_warp(uint tid, T x, OpFn op) {
#pragma unroll // TODO: find a way to make sure this loop is unrolled
    for (uint i = warpSize / 2; 1 <= i; i >>= 1) {
        uint active = __ballot_sync(0xffffffff, tid < i);
        x = op(x, __shfl_down_sync(active, x, i));
    }
    return x;
}

template <uint blockSize, typename T, typename OpFn>
__global__
void reduce_k(T *dst, T *src, uint n, OpFn op, T identity_op) {
    uint tid = threadIdx.x;
    uint i = blockIdx.x*(blockSize*2) + tid;
    uint gridSize = blockSize*2*gridDim.x;

    T x = identity_op;
    while (i < n) {
        x = op(x, op(src[i], src[i+blockSize])); i += gridSize;
    }

    x = reduce_warp<std::min(32u, blockSize)>(tid, x, op);

    if constexpr (blockSize >= 64) {
        __shared__ T sdata[32];

        if (tid % 32 == 0) {
            sdata[tid / 32] = x;
        }

        __syncthreads();

        if (tid < 32) {
            x = sdata[tid];
            x = reduce_warp<blockSize / 32>(tid, x, op);
        }
    }

    if (tid == 0)
        dst[blockIdx.x] = x;
}

template <typename T, typename OpFn>
void reduce_once(T *dst, T *src, int b, int t, int n, OpFn op, T identity_op) {
    assert(t <= 1024 && is_power_of_2(t));
    assert(t*2 <= n && n % (t*2) == 0);

#define BlockSizeCase(t) case t: reduce_k<t, T><<<b, t>>>(dst, src, n, op, identity_op); break;
    switch (t) {
        BlockSizeCase(1024)
        BlockSizeCase( 512)
        BlockSizeCase( 256)
        BlockSizeCase( 128)
        BlockSizeCase(  64)
        BlockSizeCase(  32)
        BlockSizeCase(  16)
        BlockSizeCase(   8)
        BlockSizeCase(   4)
        BlockSizeCase(   2)
        BlockSizeCase(   1)
        default: assert(false); break;
    }
#undef BlockSizeCase
}

// TODO: T in OpFn can be omitted on caller site
// ^ something like template <typename> typename OpFn> and use parameter OpFn<T>
// TODO: op should not be a parameter, bc its an empty class anyways
// TODO: for OpFn enable simply passing labmda.
// ^ to reduce instances of this template, are calls wtih identical lambdas collapsed into single instance?
template <typename T, typename OpFn>
T reduce(T *src, OpFn op, T identity_op) {
    static_assert(1 < BLOCK);
    constexpr uint r = BLOCK*2;

    T *dst;
    hipMalloc(&dst, S * sizeof(T) / r);
    T *markForFree = dst;

    uint n = S;
    while (1 < n) {
        uint b = clamp(n/r, 1u, GRID_STRIDED);
        uint t = min(BLOCK, n/2);
        reduce_once(dst, src, b, t, n, op, identity_op);
        n = b;
        std::swap(dst, src);
    }
    std::swap(dst, src);

    T result;
    hipMemcpy(&result, dst, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(markForFree);
    return result;
}

template
uint8 reduce<uint8, BinaryAndOp<uint8>>(uint8 *src, BinaryAndOp<uint8> op, uint8 identity_op);

template
int32 reduce<int32, AddOp<int32>>(int32 *src, AddOp<int32> op, int32 identity_op);
