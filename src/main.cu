#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <chrono>
#include <tuple>
#include <thread>
#include <vector>
#include <type_traits>

#include <glm/glm.hpp>

constexpr const unsigned int W = 128;
// constexpr const unsigned int W = 1<<11;
constexpr const unsigned int S = W*W;
static_assert(W % 32 == 0);
static_assert(W <= 0xffffffff/2);

constexpr const unsigned int BLOCK_SIZE = 128;
[[maybe_unused]] constexpr const unsigned int GRID_SIZE = S / BLOCK_SIZE;
[[maybe_unused]] constexpr const unsigned int GRID_SIZE_STRIDED = std::min(128u, GRID_SIZE);
static_assert(S == GRID_SIZE * BLOCK_SIZE);

using namespace glm;
using namespace std::chrono_literals;

__device__
__forceinline__
int coord_to_id(ivec2 coord, int w) {
    return coord.x + coord.y*w;
}

__device__
__forceinline__
ivec2 id_to_coord(int id, int w) {
    return ivec2(id % w, id / w);
}

template <typename T>
__global__
void transpose_naive(T *dst, T *src, int w) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int dst_id = coord_to_id(ivec2(x, y), w);
    int src_id = coord_to_id(ivec2(y, x), w);
    dst[dst_id] = src[src_id];
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_coalesced_bankconflict(T *dst, T *src) {
    __shared__ T data[BLOCK_DIM_X][BLOCK_DIM_X];

    uint w = gridDim.x * BLOCK_DIM_X;
    uint ox = threadIdx.x + blockIdx.x*BLOCK_DIM_X;
    uint oy = threadIdx.y + blockIdx.y*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        data[threadIdx.y+y][threadIdx.x] = src[ox + (oy+y) * w];

    __syncthreads();
    ox = threadIdx.x + blockIdx.y*BLOCK_DIM_X;
    oy = threadIdx.y + blockIdx.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        dst[ox + (oy+y) * w] = data[threadIdx.x][threadIdx.y+y];
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_coalesced(T *dst, T *src) {
    __shared__ T data[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint w = gridDim.x * BLOCK_DIM_X;
    uint ox = threadIdx.x + blockIdx.x*BLOCK_DIM_X;
    uint oy = threadIdx.y + blockIdx.y*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        data[threadIdx.y+y][threadIdx.x] = src[ox + (oy+y) * w];

    __syncthreads();
    ox = threadIdx.x + blockIdx.y*BLOCK_DIM_X;
    oy = threadIdx.y + blockIdx.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        dst[ox + (oy+y) * w] = data[threadIdx.x][threadIdx.y+y];
}

__device__
float Q_rsqrt(float number) {
    long i;
    float x2, y;
    const float threehalfs = 1.5F;

    x2 = number * 0.5F;
    y  = number;
    i  = * ( long * ) &y;    // evil floating point bit level hacking
    i  = 0x5f3759df - ( i >> 1 );               // what the fuck? 
    y  = * ( float * ) &i;
    y  = y * ( threehalfs - ( x2 * y * y ) );   // 1st iteration
    //y  = y * ( threehalfs - ( x2 * y * y ) );   // 2nd iteration,

    return y;
}

__device__
ivec2 index_to_tricoord(int i) {
    // int y = 1.0/Q_rsqrt(0.25f + 2.0f*i) - 0.5f;
    int y = sqrt(0.25f + 2.0f*i) - 0.5f;
    int x = i - y*(y+1)/2;
    return ivec2(x, y);
}

int tricoord_to_index(ivec2 v) {
    return v.x + v.y*(v.y+1)/2;
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_triangle(T *src, uint w) {
    uvec2 tile = index_to_tricoord(blockIdx.x);

    __shared__ T data0[BLOCK_DIM_X][BLOCK_DIM_X + 1];
    __shared__ T data1[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint ox0 = threadIdx.x + tile.x*BLOCK_DIM_X;
    uint oy0 = threadIdx.y + tile.y*BLOCK_DIM_X;
    uint ox1 = threadIdx.x + tile.y*BLOCK_DIM_X;
    uint oy1 = threadIdx.y + tile.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        data0[threadIdx.y+y][threadIdx.x] = src[ox0 + (oy0+y) * w];
        data1[threadIdx.y+y][threadIdx.x] = src[ox1 + (oy1+y) * w];
    }

    __syncthreads();

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        src[ox0 + (oy0+y) * w] = data1[threadIdx.x][threadIdx.y+y];
        src[ox1 + (oy1+y) * w] = data0[threadIdx.x][threadIdx.y+y];
    }
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_triangle_diag(T *src) {
    __shared__ T data[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint w = gridDim.x * BLOCK_DIM_X;
    uint ox = threadIdx.x + blockIdx.x*BLOCK_DIM_X;
    uint oy = threadIdx.y + blockIdx.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        data[threadIdx.y+y][threadIdx.x] = src[ox + (oy+y) * w];

    __syncthreads();

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        src[ox + (oy+y) * w] = data[threadIdx.x][threadIdx.y+y];
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_triangle_internal(T *src, uint w) {
    uvec2 tile = index_to_tricoord(blockIdx.x);
    tile.y += 1;

    __shared__ T data0[BLOCK_DIM_X][BLOCK_DIM_X + 1];
    __shared__ T data1[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint ox0 = threadIdx.x + tile.x*BLOCK_DIM_X;
    uint oy0 = threadIdx.y + tile.y*BLOCK_DIM_X;
    uint ox1 = threadIdx.x + tile.y*BLOCK_DIM_X;
    uint oy1 = threadIdx.y + tile.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        data0[threadIdx.y+y][threadIdx.x] = src[ox0 + (oy0+y) * w];
        data1[threadIdx.y+y][threadIdx.x] = src[ox1 + (oy1+y) * w];
    }

    __syncthreads();

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        src[ox0 + (oy0+y) * w] = data1[threadIdx.x][threadIdx.y+y];
        src[ox1 + (oy1+y) * w] = data0[threadIdx.x][threadIdx.y+y];
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLastError(__FILE__, __LINE__)
inline void checkLastError(const char* const file, const int line) {
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl
            << hipGetErrorString(err) << std::endl;
    }
}

template <bool Transposed, typename T>
__global__
void init_data_k(T *output) {
    for (int b = 0; b < blockDim.x; b += blockDim.y) {
        int x = threadIdx.x + blockIdx.x*blockDim.x;
        int y = threadIdx.y + blockIdx.y*blockDim.x + b;
        int i = x + y * W;
        if constexpr (Transposed)
            output[i] = x % 256;
        else
            output[i] = y % 256;
    }
}

template <bool Transposed, typename T>
void init_data(T *output) {
    init_data_k<Transposed><<<{W/32u, W/32u, 1u}, {32u, 4u, 1u}>>>(output);
}

template <typename T>
using ReduceOp = T(*)(T, T);

template <typename T>
__device__
T op_add(T a, T b) {
    return a + b;
}

__device__
uint8 op_binary_or(uint8 a, uint8 b) {
    return a && b;
}

__device__ ReduceOp<int32> p_op_add_int32 = op_add<int32>;
__device__ ReduceOp<uint8> p_op_binary_or = op_binary_or;

template <typename T>
ReduceOp<T> get_reduce_op_add() {
    ReduceOp<T> op;
    static_assert(std::is_same_v<T, int32>);
    hipMemcpyFromSymbol(&op, HIP_SYMBOL(p_op_add_int32), sizeof(ReduceOp<T>));
    return op;
}

ReduceOp<uint8> get_reduce_op_binary_and() {
    ReduceOp<uint8> op;
    hipMemcpyFromSymbol(&op, HIP_SYMBOL(p_op_binary_or), sizeof(ReduceOp<uint8>));
    return op;
}

constexpr const uint8 identity_binary_op = 1;
constexpr const int32 identity_op_add = 0;

/* begin
 * Mark Harris NVIDIA Developer Technology
 */
template <unsigned int blockSize, typename T>
__device__
void warpReduce(volatile T *sdata, unsigned int tid, ReduceOp<T> op) {
    if (blockSize >= 64) sdata[tid] = op(sdata[tid], sdata[tid + 32]);
    if (blockSize >= 32) sdata[tid] = op(sdata[tid], sdata[tid + 16]);
    if (blockSize >= 16) sdata[tid] = op(sdata[tid], sdata[tid + 8]);
    if (blockSize >= 8)  sdata[tid] = op(sdata[tid], sdata[tid + 4]);
    if (blockSize >= 4)  sdata[tid] = op(sdata[tid], sdata[tid + 2]);
    if (blockSize >= 2)  sdata[tid] = op(sdata[tid], sdata[tid + 1]);
}

template <unsigned int blockSize, typename T>
__global__
void reduce_k(T *g_odata, T *g_idata, unsigned int n, ReduceOp<T> op, T identity_op) {
    extern __shared__ /*__align__(sizeof(T))*/ uint8 sdata_[];
    T *sdata = reinterpret_cast<T*>(sdata_);
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = identity_op;
    while (i < n) { sdata[tid] = op(sdata[tid], op(g_idata[i], g_idata[i+blockSize])); i += gridSize; }
    __syncthreads();
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = op(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = op(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64)  { sdata[tid] = op(sdata[tid], sdata[tid + 64]);  } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid, op);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
/* end
 * Mark Harris NVIDIA Developer Technology
 */

constexpr inline bool is_power_of_2(int n) {
    return (n & (n-1)) == 0;
}

template <typename T>
void reduce_once(T *dst, T *src, int b, int t, int n, ReduceOp<T> op, T identity_op) {
    assert(t <= 512);
    assert(is_power_of_2(t));

#define BlockSizeCase(t) case t: reduce_k<t, T><<<b, t, t*sizeof(T)>>>(dst, src, n, op, identity_op); break;
    switch (t) {
        BlockSizeCase(512)
        BlockSizeCase(256)
        BlockSizeCase(128)
        BlockSizeCase( 64)
        BlockSizeCase( 32)
        BlockSizeCase( 16)
        BlockSizeCase(  8)
        BlockSizeCase(  4)
        BlockSizeCase(  2)
        BlockSizeCase(  1)
        default: assert(false); break;
    }
#undef sum_case
}

template <typename T>
void dump(T* data_d, uint n=S, uint w=W) {
    auto data = std::vector<T>(n);
    hipMemcpy(data.data(), data_d, n * sizeof(T), hipMemcpyDeviceToHost);
    std::cout << "\tdump" << std::endl;
    for (int i = 0; i < n; ++i) {
        if (i != 0 && i % w == 0)
            std::cout << std::endl;
        std::cout << (int64)data[i] << " ";
    }
    std::cout << std::endl;
}

template <typename T>
T reduce(T *src, ReduceOp<T> op, T identity_op) {
    T *dst;
    hipMalloc(&dst, S * sizeof(T));
    T *markForFree = dst;

    uint r = BLOCK_SIZE*2;
    uint n = S;

    while (1 < n) {
        uint b = std::max(1u, n/r);
        b = std::min(b, GRID_SIZE_STRIDED);
        uint t = std::min(BLOCK_SIZE, n/2);
        reduce_once(dst, src, b, t, n, op, identity_op);
        n = b;
        std::swap(dst, src);
    }
    std::swap(dst, src);

    T result;
    hipMemcpy(&result, dst, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(markForFree);
    return result;
}

template <typename T, typename U>
__global__
void compare_k(uint8 *dst, T *a, U *b) {
    for (int s = 0; s < blockDim.x; s += blockDim.y) {
        int x = threadIdx.x + blockIdx.x*blockDim.x;
        int y = threadIdx.y + blockIdx.y*blockDim.x + s;
        int i = x + y * W;
        dst[i] = a[i] == b[i];
    }
}

template <typename T, typename U>
bool compare(T *a, U *b) {
    uint8 *result;
    hipMalloc(&result, S * sizeof(*result));
    compare_k<<<{W/32u, W/32u, 1u}, {32u, 4u, 1u}>>>(result, a, b);

    bool result_h = reduce(result, get_reduce_op_binary_and(), identity_binary_op);
    hipFree(result);
    return result_h;
}

template <typename Fn>
auto perf(Fn fn) {
    using namespace std::chrono;

    float dt_gpu;
    hipEvent_t start_event, end_event;
    hipEventCreate(&start_event);
    hipEventCreate(&end_event);

    hipDeviceSynchronize();
    auto start = steady_clock::now();

    hipEventRecord(start_event);
    fn();
    hipEventRecord(end_event);

    hipEventSynchronize(end_event);
    auto end = steady_clock::now();

    hipDeviceSynchronize();
    hipEventElapsedTime(&dt_gpu, start_event, end_event);
    hipEventDestroy(end_event);
    hipEventDestroy(start_event);

    float dt_cpu = duration_cast<microseconds>(end - start).count() / 1000.0f;
    return std::make_tuple(dt_cpu, dt_gpu);
}

template <typename T, typename Fn>
std::string experiment(uint8 *expected, uvec2 c, Fn fn) {
    T *input, *output;
    hipMalloc(&input, S * sizeof(*input));
    hipMalloc(&output, S * sizeof(*output));
    init_data<false>(input);

    auto [dt_cpu, dt_gpu] = perf([&] {
        fn(c, output, input);
    });
    CHECK_LAST_CUDA_ERROR();

    auto ok = compare(expected, output);

    hipFree(input);
    hipFree(output);

    std::stringstream ss;
    ss
        << "b=" << sizeof(T)
        << "\tdt_cpu=" << dt_cpu
        << "\tdt_gpu=" << dt_gpu
        << std::boolalpha << "\t ok=" << ok
        << std::endl;
    return ss.str();
}

template <typename Fn>
std::string run_experiments_for_element_sizes(uint8 *expected, uvec2 c, Fn fn) {
    std::stringstream ss;
    ss
        << experiment<uint8> (expected, c, fn)
        << experiment<uint16>(expected, c, fn)
        << experiment<uint32>(expected, c, fn)
        << experiment<uint64>(expected, c, fn);
    return ss.str();
}

using InitRangePred = int8(*)(uint);

__device__
int8 pred_one(uint i) {
    return 1;
}

__device__
InitRangePred p_pred_one = pred_one;

template <typename T>
__global__
void init_range(T *dst, InitRangePred pred) {
    uint i = threadIdx.x + blockIdx.x*blockDim.x;
    dst[i] = pred(i);
}

InitRangePred get_pred_one() {
    InitRangePred pred;
    hipMemcpyFromSymbol(&pred, HIP_SYMBOL(p_pred_one), sizeof(InitRangePred));
    return pred;
}

int main()
{
    std::this_thread::sleep_for(50ms);
    std::stringstream ss;
    ss << std::boolalpha;

    uint8 *expected;
    hipMalloc(&expected, S * sizeof(*expected));
    init_data<true>(expected);

    {
        ss << "\tlets first test the helper functions" << std::endl;
        {
            int32 *data;
            hipMalloc(&data, S * sizeof(*data));
            init_range<<<GRID_SIZE, BLOCK_SIZE>>>(data, get_pred_one());
            auto sum = reduce(data, get_reduce_op_add<int32>(), identity_op_add);
            hipFree(data);

            ss
                << "reduce(op=add)       ok=" << (sum == S) << std::endl;
        }
        {
            uint8 *data;
            hipMalloc(&data, S * sizeof(*data));
            init_range<<<GRID_SIZE, BLOCK_SIZE>>>(data, get_pred_one());
            auto result_true = reduce(data, get_reduce_op_binary_and(), identity_binary_op);

            init_range<<<GRID_SIZE, BLOCK_SIZE>>>(data, get_pred_one());
            uint8 zero = 0;
            hipMemcpy(data, &zero, sizeof(uint8), hipMemcpyHostToDevice);
            auto result_false = reduce(data, get_reduce_op_binary_and(), identity_binary_op);
            hipFree(data);

            ss
                << "reduce(op=binary_or) ok=" << static_cast<bool>(result_true) << std::endl
                << "reduce(op=binary_or) ok=" << !static_cast<bool>(result_false) << std::endl;
        }
        {
            uint8 *data, *data2, *data_transposed;
            hipMalloc(&data, S * sizeof(*data));
            hipMalloc(&data2, S * sizeof(*data2));
            hipMalloc(&data_transposed, S * sizeof(*data_transposed));
            init_data<false>(data);
            init_data<false>(data2);
            init_data<true>(data_transposed);
            uint8 one = 1;
            hipMemcpy(data2, &one, sizeof(uint8), hipMemcpyHostToDevice);
            auto result_true = compare(data, data);
            auto result_false = compare(data, data_transposed);
            auto result_false2 = compare(data, data2);
            hipFree(data);
            hipFree(data2);
            hipFree(data_transposed);

            ss
                << "compare(equal)       ok=" << static_cast<bool>(result_true) << std::endl
                << "compare(different)   ok=" << !static_cast<bool>(result_false) << std::endl
                << "compare(different)   ok=" << !static_cast<bool>(result_false2) << std::endl;
        }

        ss << std::endl;
    }

    ss
        << "width: " << W << std::endl
        << "pixels: " << S << std::endl
        << "dt (delta time) in milliseconds" << std::endl
        << std::endl;

    [[maybe_unused]] auto fn_0 = [] <typename T> (uvec2 c, T *output, T *input) {
        transpose_naive<<<{W/c.x, W/c.y, 1u}, {c.x, c.y, 1u}>>>(output, input, W);
    };

    [[maybe_unused]] auto fn_1 = [] <typename T> (uvec2 c, T *output, T *input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        transpose_coalesced_bankconflict<X, Y><<<{W/X, W/X, 1u}, {X, Y, 1u}>>>(output, input);
    };

    [[maybe_unused]] auto fn_2 = [] <typename T> (uvec2 c, T *output, T *input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        transpose_coalesced<X, Y><<<{W/X, W/X, 1u}, {X, Y, 1u}>>>(output, input);
    };

    [[maybe_unused]] auto fn_3 = [] <typename T> (uvec2 c, T *&output, T *&input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        uint B = tricoord_to_index(ivec2(W/X - 1, W/X - 1)) + 1;
        transpose_triangle<X, Y><<<{B, 1u, 1u}, {X, Y, 1u}>>>(input, W);
        std::swap(output, input);
    };

    [[maybe_unused]] auto fn_4 = [] <typename T> (uvec2 c, T *&output, T *&input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        uint B = tricoord_to_index(ivec2(W/X - 2, W/X - 2)) + 1;
        transpose_triangle_internal<X, Y><<<{B, 1u, 1u}, {X, Y, 1u}>>>(input, W);
        transpose_triangle_diag<X, Y><<<{W/X, 1u, 1u}, {X, Y, 1u}>>>(input);
        std::swap(output, input);
    };

    ss
        << "transpose_naive" << std::endl
        << run_experiments_for_element_sizes(expected, {4, 32}, fn_0)
        << std::endl

        << "transpose_coalesced_bankconflict" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_1)
        << std::endl

        << "transpose_coalesced" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_2)
        << std::endl

        << "transpose_triangle" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_3)
        << std::endl

        << "transpose_triangle separate diag&internal" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_4)
        << std::endl
    ;

    hipFree(expected);
    std::cout << ss.str();

    return 0;
}
