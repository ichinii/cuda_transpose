#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <chrono>
#include <tuple>
#include <thread>
#include <vector>
#include <type_traits>

#include <glm/glm.hpp>

using namespace glm;
using namespace std::chrono_literals;

// constexpr const unsigned int W = 128;
constexpr const uint W = 1<<12;
constexpr const uint S = W*W;
static_assert(W % 32 == 0);
static_assert(W <= 1<<16);

constexpr const unsigned int BLOCK_SIZE = 128;
[[maybe_unused]] constexpr const unsigned int GRID_SIZE = S / BLOCK_SIZE;
[[maybe_unused]] constexpr const unsigned int GRID_SIZE_STRIDED = std::min(128u, GRID_SIZE);
static_assert(S == GRID_SIZE * BLOCK_SIZE);

__device__
__forceinline__
int coord_to_id(ivec2 coord, int w) {
    return coord.x + coord.y*w;
}

__device__
__forceinline__
ivec2 id_to_coord(int id, int w) {
    return ivec2(id % w, id / w);
}

template <typename T>
__global__
void transpose_naive(T *dst, T *src, int w) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int dst_id = coord_to_id(ivec2(x, y), w);
    int src_id = coord_to_id(ivec2(y, x), w);
    dst[dst_id] = src[src_id];
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_coalesced_bankconflict(T *dst, T *src) {
    __shared__ T data[BLOCK_DIM_X][BLOCK_DIM_X];

    uint w = gridDim.x * BLOCK_DIM_X;
    uint ox = threadIdx.x + blockIdx.x*BLOCK_DIM_X;
    uint oy = threadIdx.y + blockIdx.y*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        data[threadIdx.y+y][threadIdx.x] = src[ox + (oy+y) * w];

    __syncthreads();
    ox = threadIdx.x + blockIdx.y*BLOCK_DIM_X;
    oy = threadIdx.y + blockIdx.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        dst[ox + (oy+y) * w] = data[threadIdx.x][threadIdx.y+y];
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_coalesced(T *dst, T *src) {
    __shared__ T data[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint w = gridDim.x * BLOCK_DIM_X;
    uint ox = threadIdx.x + blockIdx.x*BLOCK_DIM_X;
    uint oy = threadIdx.y + blockIdx.y*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        data[threadIdx.y+y][threadIdx.x] = src[ox + (oy+y) * w];

    __syncthreads();
    ox = threadIdx.x + blockIdx.y*BLOCK_DIM_X;
    oy = threadIdx.y + blockIdx.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        dst[ox + (oy+y) * w] = data[threadIdx.x][threadIdx.y+y];
}

__device__
float Q_rsqrt(float number) {
    long i;
    float x2, y;
    const float threehalfs = 1.5F;

    x2 = number * 0.5F;
    y  = number;
    i  = * ( long * ) &y;    // evil floating point bit level hacking
    i  = 0x5f3759df - ( i >> 1 );               // what the fuck? 
    y  = * ( float * ) &i;
    y  = y * ( threehalfs - ( x2 * y * y ) );   // 1st iteration
    // y  = y * ( threehalfs - ( x2 * y * y ) );   // 2nd iteration,

    return y;
}

__device__
ivec2 index_to_tricoord(int i) {
    // int y = 1.0/Q_rsqrt(0.25f + 2.0f*i) - 0.5f;
    int y = sqrt(0.25f + 2.0f*i) - 0.5f;
    int x = i - y*(y+1)/2;
    return ivec2(x, y);
}

int tricoord_to_index(ivec2 v) {
    return v.x + v.y*(v.y+1)/2;
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_triangle(T *src, uint w) {
    uvec2 tile = index_to_tricoord(blockIdx.x);

    __shared__ T data0[BLOCK_DIM_X][BLOCK_DIM_X + 1];
    __shared__ T data1[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint ox0 = threadIdx.x + tile.x*BLOCK_DIM_X;
    uint oy0 = threadIdx.y + tile.y*BLOCK_DIM_X;
    uint ox1 = threadIdx.x + tile.y*BLOCK_DIM_X;
    uint oy1 = threadIdx.y + tile.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        data0[threadIdx.y+y][threadIdx.x] = src[ox0 + (oy0+y) * w];
        data1[threadIdx.y+y][threadIdx.x] = src[ox1 + (oy1+y) * w];
    }

    __syncthreads();

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        src[ox0 + (oy0+y) * w] = data1[threadIdx.x][threadIdx.y+y];
        src[ox1 + (oy1+y) * w] = data0[threadIdx.x][threadIdx.y+y];
    }
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_triangle_diag(T *src) {
    __shared__ T data[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint w = gridDim.x * BLOCK_DIM_X;
    uint ox = threadIdx.x + blockIdx.x*BLOCK_DIM_X;
    uint oy = threadIdx.y + blockIdx.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        data[threadIdx.y+y][threadIdx.x] = src[ox + (oy+y) * w];

    __syncthreads();

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y)
        src[ox + (oy+y) * w] = data[threadIdx.x][threadIdx.y+y];
}

template <uint BLOCK_DIM_X, uint BLOCK_DIM_Y, typename T>
__global__
void transpose_triangle_internal(T *src, uint w) {
    uvec2 tile = index_to_tricoord(blockIdx.x);
    tile.y += 1;

    __shared__ T data0[BLOCK_DIM_X][BLOCK_DIM_X + 1];
    __shared__ T data1[BLOCK_DIM_X][BLOCK_DIM_X + 1];

    uint ox0 = threadIdx.x + tile.x*BLOCK_DIM_X;
    uint oy0 = threadIdx.y + tile.y*BLOCK_DIM_X;
    uint ox1 = threadIdx.x + tile.y*BLOCK_DIM_X;
    uint oy1 = threadIdx.y + tile.x*BLOCK_DIM_X;

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        data0[threadIdx.y+y][threadIdx.x] = src[ox0 + (oy0+y) * w];
        data1[threadIdx.y+y][threadIdx.x] = src[ox1 + (oy1+y) * w];
    }

    __syncthreads();

    for (uint y = 0; y < BLOCK_DIM_X; y += BLOCK_DIM_Y) {
        src[ox0 + (oy0+y) * w] = data1[threadIdx.x][threadIdx.y+y];
        src[ox1 + (oy1+y) * w] = data0[threadIdx.x][threadIdx.y+y];
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLastError(__FILE__, __LINE__)
inline void checkLastError(const char* const file, const int line) {
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl
            << hipGetErrorString(err) << std::endl;
    }
}

template <typename T>
void dump(T* data_d, uint n=S, uint w=W) {
    auto data = std::vector<T>(n);
    hipMemcpy(data.data(), data_d, n * sizeof(T), hipMemcpyDeviceToHost);
    std::cout << "\tdump" << std::endl;
    for (int i = 0; i < n; ++i) {
        if (i != 0 && i % w == 0)
            std::cout << std::endl;
        std::cout << (int64)data[i] << " ";
    }
    std::cout << std::endl;
}

template <bool Transposed, typename T>
__global__
void init_data_k(T *output) {
    for (int b = 0; b < blockDim.x; b += blockDim.y) {
        int x = threadIdx.x + blockIdx.x*blockDim.x;
        int y = threadIdx.y + blockIdx.y*blockDim.x + b;
        int i = x + y * W;
        if constexpr (Transposed)
            output[i] = x % 256;
        else
            output[i] = y % 256;
    }
}

template <bool Transposed, typename T>
void init_data(T *output) {
    init_data_k<Transposed><<<{W/32u, W/32u, 1u}, {32u, 4u, 1u}>>>(output);
}

template <typename T>
struct AdditionOp {
    __device__ __host__
    __forceinline__
    T operator()(T a, T b) { return a + b; }
};

template <typename T>
struct BinaryAndOp {
    __device__ __host__
    __forceinline__
    T operator()(T a, T b) { return a && b; }
};

constexpr inline bool is_power_of_2(int n) {
    return (n & (n-1)) == 0;
}

template <unsigned int blockSize, typename T, typename OpFn>
__device__
__forceinline__
T reduce_warp(T x, OpFn op) {
#pragma unroll // TODO: find a way to make sure this loop is unrolled
    for (int i = std::min(32u, blockSize); 2 <= i; i /= 2)
        x = op(x, __shfl_down_sync(0xffffffff, x, i/2));
    return x;
}

template <unsigned int blockSize, typename T, typename OpFn>
__global__
void reduce_k(T *g_odata, T *g_idata, unsigned int n, OpFn op, T identity_op) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T x = identity_op;
    while (i < n) { x = op(x, op(g_idata[i], g_idata[i+blockSize])); i += gridSize; }
    __syncthreads();

    x = reduce_warp<blockSize>(x, op);

    if constexpr (blockSize >= 64) {
        __shared__ T sdata[blockSize / 32];

        if (tid % 32 == 0) {
            sdata[tid / 32] = x;
        }

        __syncthreads();

        if (tid < blockSize / 32) {
            x = sdata[tid];
            x = reduce_warp<blockSize / 32>(x, op);
        }
    }

    if (tid == 0)
        g_odata[blockIdx.x] = x;
}

template <typename T, typename OpFn>
void reduce_once(T *dst, T *src, int b, int t, int n, OpFn op, T identity_op) {
    assert(t <= 1024);
    assert(is_power_of_2(t));
    assert(n % (t*2) == 0);

#define BlockSizeCase(t) case t: reduce_k<t, T><<<b, t>>>(dst, src, n, op, identity_op); break;
    switch (t) {
        BlockSizeCase(1024)
        BlockSizeCase( 512)
        BlockSizeCase( 256)
        BlockSizeCase( 128)
        BlockSizeCase(  64)
        BlockSizeCase(  32)
        BlockSizeCase(  16)
        BlockSizeCase(   8)
        BlockSizeCase(   4)
        BlockSizeCase(   2)
        BlockSizeCase(   1)
        default: assert(false); break;
    }
#undef BlockSizeCase
}

// TODO: T in OpFn can be omitted on caller site
// ^ something like template <typename> typename OpFn> and use parameter OpFn<T>
// TODO: op should not be a parameter, bc its an empty class anyways
// TODO: for OpFn enable simply passing labmda.
// ^ to reduce instances of this template, are calls wtih identical lambdas collapsed into single instance?
template <typename T, typename OpFn>
T reduce(T *src, OpFn op, T identity_op) {
    T *dst;
    hipMalloc(&dst, S * sizeof(T));
    T *markForFree = dst;

    uint r = BLOCK_SIZE*2;
    uint n = S;

    while (1 < n) {
        uint b = std::max(1u, n/r);
        b = std::min(b, GRID_SIZE_STRIDED);
        uint t = std::min(BLOCK_SIZE, n/2);
        reduce_once(dst, src, b, t, n, op, identity_op);
        n = b;
        std::swap(dst, src);
    }
    std::swap(dst, src);

    T result;
    hipMemcpy(&result, dst, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(markForFree);
    return result;
}

template <typename T, typename U>
__global__
void compare_k(uint8 *dst, T *a, U *b) {
    for (int s = 0; s < blockDim.x; s += blockDim.y) {
        int x = threadIdx.x + blockIdx.x*blockDim.x;
        int y = threadIdx.y + blockIdx.y*blockDim.x + s;
        int i = x + y * W;
        dst[i] = a[i] == b[i];
    }
}

template <typename T, typename U>
bool compare(T *a, U *b) {
    uint8 *result;
    hipMalloc(&result, S * sizeof(*result));
    compare_k<<<{W/32u, W/32u, 1u}, {32u, 4u, 1u}>>>(result, a, b);

    bool result_h = reduce(result, BinaryAndOp<uint8>(), static_cast<uint8>(1));
    hipFree(result);
    return result_h;
}

template <typename Fn>
auto perf(Fn fn) {
    using namespace std::chrono;

    float dt_gpu;
    hipEvent_t start_event, end_event;
    hipEventCreate(&start_event);
    hipEventCreate(&end_event);

    hipDeviceSynchronize();
    auto start = steady_clock::now();

    hipEventRecord(start_event);
    fn();
    hipEventRecord(end_event);

    hipEventSynchronize(end_event);
    auto end = steady_clock::now();

    hipDeviceSynchronize();
    hipEventElapsedTime(&dt_gpu, start_event, end_event);
    hipEventDestroy(end_event);
    hipEventDestroy(start_event);

    float dt_cpu = duration_cast<microseconds>(end - start).count() / 1000.0f;
    return std::make_tuple(dt_cpu, dt_gpu);
}

template <typename T, typename Fn>
std::string experiment(uint8 *expected, uvec2 c, Fn fn) {
    T *input, *output;
    hipMalloc(&input, S * sizeof(*input));
    hipMalloc(&output, S * sizeof(*output));
    init_data<false>(input);

    auto [dt_cpu, dt_gpu] = perf([&] {
        fn(c, output, input);
    });
    CHECK_LAST_CUDA_ERROR();

    auto ok = compare(expected, output);

    hipFree(input);
    hipFree(output);

    std::stringstream ss;
    ss
        << "b=" << sizeof(T)
        << "\tdt_cpu=" << dt_cpu
        << "\tdt_gpu=" << dt_gpu
        << std::boolalpha << "\t ok=" << ok
        << std::endl;
    return ss.str();
}

template <typename Fn>
std::string run_experiments_for_element_sizes(uint8 *expected, uvec2 c, Fn fn) {
    std::stringstream ss;
    ss
        << experiment<uint8> (expected, c, fn)
        << experiment<uint16>(expected, c, fn)
        << experiment<uint32>(expected, c, fn)
        << experiment<uint64>(expected, c, fn)
    ;
    return ss.str();
}

template <typename T>
__global__
void init_range_with_one(T *dst) {
    uint i = threadIdx.x + blockIdx.x*blockDim.x;
    dst[i] = 1;
}

int main()
{
    std::this_thread::sleep_for(50ms);
    std::stringstream ss;
    ss << std::boolalpha;

    uint8 *expected;
    hipMalloc(&expected, S * sizeof(*expected));
    init_data<true>(expected);

    if constexpr (true) {
        ss << "lets first test the helper functions" << std::endl;
        {
            int32 *data;
            hipMalloc(&data, S * sizeof(*data));
            init_range_with_one<<<GRID_SIZE, BLOCK_SIZE>>>(data);
            int32 sum = reduce(data, AdditionOp<int32>(), 0);
            hipFree(data);

            ss
                << "reduce(op=add)        ok=" << (sum == S) << std::endl;
        }
        {
            uint8 *data;
            hipMalloc(&data, S * sizeof(*data));
            init_range_with_one<<<GRID_SIZE, BLOCK_SIZE>>>(data);
            auto result_true = reduce(data, BinaryAndOp<uint8>(), static_cast<uint8>(1));

            init_range_with_one<<<GRID_SIZE, BLOCK_SIZE>>>(data);
            uint8 zero = 0;
            hipMemcpy(data, &zero, sizeof(uint8), hipMemcpyHostToDevice);
            auto result_false = reduce(data, BinaryAndOp<uint8>(), static_cast<uint8>(1));
            hipFree(data);

            ss
                << "reduce(op=binary_and) ok=" << static_cast<bool>(result_true) << std::endl
                << "reduce(op=binary_and) ok=" << !static_cast<bool>(result_false) << std::endl;
        }
        {
            uint8 *data, *data2, *data_transposed;
            hipMalloc(&data, S * sizeof(*data));
            hipMalloc(&data2, S * sizeof(*data2));
            hipMalloc(&data_transposed, S * sizeof(*data_transposed));
            init_data<false>(data);
            init_data<false>(data2);
            init_data<true>(data_transposed);
            uint8 one = 1;
            hipMemcpy(data2, &one, sizeof(uint8), hipMemcpyHostToDevice);
            auto result_true = compare(data, data);
            auto result_false = compare(data, data_transposed);
            auto result_false2 = compare(data, data2);
            hipFree(data);
            hipFree(data2);
            hipFree(data_transposed);

            ss
                << "compare(equal)        ok=" << static_cast<bool>(result_true) << std::endl
                << "compare(different)    ok=" << !static_cast<bool>(result_false) << std::endl
                << "compare(different)    ok=" << !static_cast<bool>(result_false2) << std::endl;
        }
        ss << std::endl;
    }

    ss
        << "width: " << W << std::endl
        << "pixels: " << S << std::endl
        << "dt (delta time) in milliseconds" << std::endl
        << std::endl;

    [[maybe_unused]] auto fn_0 = [] <typename T> (uvec2 c, T *output, T *input) {
        transpose_naive<<<{W/c.x, W/c.y, 1u}, {c.x, c.y, 1u}>>>(output, input, W);
    };

    [[maybe_unused]] auto fn_1 = [] <typename T> (uvec2 c, T *output, T *input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        transpose_coalesced_bankconflict<X, Y><<<{W/X, W/X, 1u}, {X, Y, 1u}>>>(output, input);
    };

    [[maybe_unused]] auto fn_2 = [] <typename T> (uvec2 c, T *output, T *input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        transpose_coalesced<X, Y><<<{W/X, W/X, 1u}, {X, Y, 1u}>>>(output, input);
    };

    [[maybe_unused]] auto fn_3 = [] <typename T> (uvec2 c, T *&output, T *&input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        uint B = tricoord_to_index(ivec2(W/X - 1, W/X - 1)) + 1;
        transpose_triangle<X, Y><<<{B, 1u, 1u}, {X, Y, 1u}>>>(input, W);
        std::swap(output, input);
    };

    [[maybe_unused]] auto fn_4 = [] <typename T> (uvec2 c, T *&output, T *&input) {
        constexpr const uint X = 32;
        constexpr const uint Y = 4;
        uint B = tricoord_to_index(ivec2(W/X - 2, W/X - 2)) + 1;
        transpose_triangle_internal<X, Y><<<{B, 1u, 1u}, {X, Y, 1u}>>>(input, W);
        transpose_triangle_diag<X, Y><<<{W/X, 1u, 1u}, {X, Y, 1u}>>>(input);
        std::swap(output, input);
    };

    ss
        << "transpose_naive" << std::endl
        << run_experiments_for_element_sizes(expected, {4, 32}, fn_0)
        << std::endl

        << "transpose_coalesced_bankconflict" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_1)
        << std::endl

        << "transpose_coalesced" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_2)
        << std::endl

        << "transpose_triangle" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_3)
        << std::endl

        << "transpose_triangle separate diag&internal" << std::endl
        << run_experiments_for_element_sizes(expected, {}, fn_4)
        << std::endl
    ;

    hipFree(expected);
    std::cout << ss.str();

    return 0;
}
